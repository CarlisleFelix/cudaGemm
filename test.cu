#include <iostream>
#include "./header/utils.cuh"
// #include "./kernel/kernel.cuh"

#define M 3  // A 的行数
#define K 3  // A 的列数 / B 的行数
#define N 3  // B 的列数

void test1(){
    // 行主序矩阵 A 和 B
    float A[M * K] = {3, 2, 1, 4, 5, 6, 7, 8, 9};  // 3x4
    float B[K * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};  // 4x2
    float C[M * N] = {0};  // 3x2 结果矩阵

    float alpha = 1.0f;
    float beta = 0.0f;

    float *d_A, *d_B, *d_C;

    // 初始化CUDA并分配设备内存
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    // 将数据从主机拷贝到设备
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    testKernel(0, M, N, K, alpha, d_A, d_B, beta, d_C, handle);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Result matrix C:" << std::endl;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            // std::cout << C[i + M * j] << " ";
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);
}

// #define CEIL_DIV(a,b) ((a) + (b - 1) / (b))

void test2(){
    // 行主序矩阵 A 和 B
    float A[M * K] = {3, 2, 1, 4, 5, 6, 7, 8, 9};  // 3x4
    float B[K * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};  // 4x2
    float C[M * N] = {0};  // 3x2 结果矩阵

    float alpha = 1.0f;
    float beta = 0.0f;

    float *d_A, *d_B, *d_C;

    // 初始化CUDA并分配设备内存
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    // 将数据从主机拷贝到设备
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice);

    // dim3 blockDim(32, 32);
    // dim3 gridDim(1, 1);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    testKernel(1, M, N, K, alpha, d_A, d_B, beta, d_C, handle);

    hipDeviceSynchronize();

    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result matrix C:" << std::endl;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            // std::cout << C[i + M * j] << " ";
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}


int main() {
    test1();
    test2();
    return 0;
}